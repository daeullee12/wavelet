#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
// #include "../src/cuda_ptr.h"
#include "utils.h"
#include "error.h"
#include <iostream>
// #include <studio.h>


// 1.0/ sqrt(2)
#define haar 0.5f

/*  Haar wavelets forward horizontal and vertical passes 
    To get the full decomposition we apply one after the other
    log_2(N) times and its done */

template<typename T>
__global__ void gpu_haar_horizontal(T* in, const int n, T* out, const int N)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if( i < n && j < n/2)
	{
		int idx_in 	= i*N + 2*j;    // (i,2*j)
		int idx_out 	= j + i*N;      // (i,j)

		out[idx_out] 		= haar*(in[idx_in] + in[idx_in+1]);
        // out(i,2*j + n/2)
		out[idx_out + n/2] 	= haar*(in[idx_in] - in[idx_in+1]);
	}
}

template<typename T>
__global__ void gpu_haar_vertical(T* in, const int n, T* out, const int N)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if( i < n/2 && j < n)
	{
		int in_idx_1 	= 2*i*N + j;
		int in_idx_2 	= (2*i+1)*N + j;
		int out_idx 	= j + i*N;

        out[out_idx]            = haar*(in[in_idx_1] + in[in_idx_2]);
        // out(i+n/2,j)
        out[out_idx + N*n/2]    = haar*(in[in_idx_1] - in[in_idx_2]);
	}
}

template<typename T>
__global__ void gpu_low_pass(T* x, const int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if(i < n && j < n)
    {
        if(fabs(x[i*n+j]) < 1.5f)
        {
            x[i*n+j] = 0.0f;
        }
    }
}

void mat_to_float(unsigned char* in, float* out, int width, int height)
{
    for(int i = 0; i < height; i++)
    {
        for(int j = 0; j < width; j++)
        {
            out[i * width + j] = static_cast<float>(in[i * width + j]);
        }
    }
}

void float_to_mat(float *in, unsigned char* out, int width, int height)
{
    for(int i = 0; i < height; i++)
    {
        for(int j = 0; j < width; j++)
        {
            out[i * width + j] = static_cast<unsigned char>(fabs(in[i * width + j]));
        }
    }
}

//define the haar wavelet transform
void run_haar_wavelet_gpu(float *channel_img, int width, int height, int haar_level);
{
    int N = width * height;
    assert(check_power_two(N));

    size_t size = N * sizeof(float);
    float *d_src, *d_dst;
    int n = N;

    int threadsPerBlock = 16;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    HANDLE_ERROR(hipMalloc((void**)&d_src, size));
    HANDLE_ERROR(hipMalloc((void**)&d_dst, size));

    HANDLE_ERROR(hipMemcpy(d_src, channel_img, size, hipMemcpyHostToDevice));

    clock_t begin, end;
    begin = clock();

    dim3 threads(threadsPerBlock, threadsPerBlock);
    dim3 blocks(blocksPerGrid, blocksPerGrid);

    for (int level = 0; level < haar_level; ++level)
    {
        blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock; // Update blocksPerGrid for each level
        gpu_haar_horizontal<<<blocks, threads>>>(d_src, n, d_dst, width);
        gpu_low_pass<<<blocks, threads>>>(d_dst, n);
        gpu_haar_vertical<<<blocks, threads>>>(d_dst, n, d_src, width);
        gpu_low_pass<<<blocks, threads>>>(d_src, n);
        n = n >> 1;
    }

    hipDeviceSynchronize();
    end = clock();
    HANDLE_ERROR(hipMemcpy(channel_img, d_src, size, hipMemcpyDeviceToHost));

    printf("GPU Elapsed: %lfs \n", elapsed(begin, end));

    hipFree(d_src);
    hipFree(d_dst);
}


