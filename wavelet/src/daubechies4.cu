#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "daubechies4.cuh"
// #include "utils.h"
#include "error.h"

// Ensure no conflicting identifiers with the `numbers` header
#undef numbers

/*  The Daubechies-4 wavelet forward pass
    I adapted this code from http://bearcave.com/misl/misl_tech/wavelets/index.html
    To compute the full wavelet transform of a signal of size N
    We call this kernel log_2(N) times (assuming N is power of 2) */

// Define device constants
__constant__ double g[4];
__constant__ double h[4];
__constant__ double ig[4];
__constant__ double ih[4];

// __host__ __device__
inline double elapsed(clock_t start, clock_t end)
{
    return double(end - start) / CLOCKS_PER_SEC;
}
// __host__ __device__
inline bool check_power_two(int x)
{
    return (x & (x - 1)) == 0;
}


__global__ void gpu_dwt_pass(float *src, float *dest, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int half = n >> 1;

    if(2*i < (n-3)) {
        dest[i]             = src[2*i]*h[0] + src[2*i+1]*h[1] + src[2*i+2]*h[2] + src[2*i+3]*h[3];
        dest[i+half]        = src[2*i]*g[0] + src[2*i+1]*g[1] + src[2*i+2]*g[2] + src[2*i+3]*g[3];
    }
    if(2*i == (n-2)) {
        dest[i]         = src[n-2]*h[0] + src[n-1]*h[1] + src[0]*h[2] + src[1]*h[3];
        dest[i+half]    = src[n-2]*g[0] + src[n-1]*g[1] + src[0]*g[2] + src[1]*g[3];
    }
}

void run_daubechies4_wavelet_gpu(float *channel_img, int width, int height, int stride)
{
    int N = width * height;
    assert(check_power_two(N));

    size_t size = N * sizeof(float);
    float *d_src, *d_dst;
    int n = N;

    int threadsPerBlock = 512;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    HANDLE_ERROR(hipMalloc((void**)&d_src, size));
    HANDLE_ERROR(hipMalloc((void**)&d_dst, size));

    HANDLE_ERROR(hipMemcpy(d_src, channel_img, size, hipMemcpyHostToDevice));

    clock_t begin, end;

    begin = clock();
    while (n >= 4)
    {
        gpu_dwt_pass<<<blocksPerGrid, threadsPerBlock>>>(d_src, d_dst, n);
        // we need only copy the n first elements, not the whole signal
        HANDLE_ERROR(hipMemcpy(d_src, d_dst, n * sizeof(float), hipMemcpyDeviceToDevice));
        n = n >> 1;
    }
    hipDeviceSynchronize();
    end = clock();
    HANDLE_ERROR(hipMemcpy(channel_img, d_src, size, hipMemcpyDeviceToHost));

    printf("GPU Elapsed: %lfs \n", elapsed(begin, end));

    hipFree(d_src);
    hipFree(d_dst);

}
